#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//---------------------------------------------------------------|
//
// La funcion "__global__" se conoce como kernel. 
//
// Solo puede ser de tipo "void".
//
// Las funciones "__global__" se ejecutan en la GPU, pero se 
// llaman desde la CPU.(No desde funciones ejecutadas en la GPU) 
//
// Cada llamada a esta funcion, lanza multiples hilos en paralelo.
//
//---------------------------------------------------------------|


__global__ void sumaVectores(float *A, float *B, float *C, int n){

   int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calcula el indice global unico del hilo dentro de una cuadricula unidimensional.

// blockIdx.x -> indice del bloque actual en la cuadricula. En este caso, identifica la posicion del bloque actual en el eje X.

// blockDim.x -> numero de hilos por bloque en el eje X.

// threadIdx.x -> indice del hilo dentro del bloque actual.  


  if (idx < n){

	C[idx] = A[idx] + B[idx];
	
 }  


}

int main(int argc, char **argv){

	int n = 1000;


	//Memoria CPU
	float *h_A = (float *)malloc(n * sizeof(float));
	float *h_B = (float *)malloc(n * sizeof(float));
	float *h_C = (float *)malloc(n * sizeof(float));



for(int i = 0; i < n; i++){

	h_A[i] = i * 1.0;
	h_B[i] = i * 2.0;
	
}



//--------------------------------------------------------------------------%
//Memoria GPU

float *d_A,*d_B,*d_C;

	hipMalloc((void**)&d_A,n * sizeof(float)); // Reserva memoria en la GPU.
	hipMalloc((void**)&d_B,n * sizeof(float));
	hipMalloc((void**)&d_C,n * sizeof(float));


hipMemcpy( d_A, h_A, n * sizeof(float), hipMemcpyHostToDevice); // Copia datos entre la CPU y la GPU.
hipMemcpy( d_B, h_B, n * sizeof(float), hipMemcpyHostToDevice); // "cudaMemcpyHostToDevice" -> copia desde la memoria del host (CPU) a la memoria del dispositivo (GPU).

/*

   Otros tipos:

   "cudaMemcpyDeviceToHost" -> copia desde la memoria del dispositivo (GPU) a la memoria del host (CPU).

   "cudaMemcpyDeviceToDevice" -> copia dentro de la memoria del dispositivo. (de una region a otra)

   "cudaMemcpyHostToHost" -> copia dentro de la memoria del host. ( de una region a otra)


*/

//--------------------------------------------------------------------------%
//Kernel

int nb = (n + 256 - 1)/256;


sumaVectores<<<nb, 256>>>( d_A, d_B, d_C, n); // Lanza un kernel en la GPU.

// nb = numero de bloques que se lanzan | 256 = numero de hilos por bloque


// Lo que hay entre parentesis, son las mismas variables del "__global__".



//cudaDeviceSynchronize(); // Asegura que todas las operaciones en la GPU se hayan completado antes de que el host continue con la ejecucion.


hipMemcpy(h_C, d_C, n*sizeof(float), hipMemcpyDeviceToHost);


printf("\nResultado:\n");
for(int i = 0; i <  10; i++){

	printf("C[%d] = %f\n",i,h_C[i]);

}


free(h_A);
free(h_B);
free(h_C);

hipFree(d_A);
hipFree(d_B);
hipFree(d_C);



return 0;
}
