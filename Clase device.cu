#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__device__ int suma(int a, int b){ 	// "__device__" se utiliza para declarar variables o funciones que se ejecutan dentro de la GPU.
					// Solo se puede acceder a un "__device" desde el codigo que tambien se ejecute en la GPU, como el "__global__" u otro "__device__".

	return a + b;

}



__global__ void kernel(){

 	int resultado = suma(5, 3);
	
	printf("\n\nEl resultado de la suma es: %d\n", resultado);

}



int main(int argc, char **argv){

	kernel<<<1, 1>>>();

	hipDeviceSynchronize();	

	return 0;
}
