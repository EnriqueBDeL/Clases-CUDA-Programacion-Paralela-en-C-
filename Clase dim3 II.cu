#include <stdio.h>
#include <hip/hip_runtime.h>

//------------------------------------------------|
// Contenido: sentencia dim3 con dos dimensiones.
//------------------------------------------------|

__global__ void restarMatrices(int *A, int *B, int *C, int filas, int columnas) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < columnas && idy < filas) {
        int index = idy * columnas + idx;
        C[index] = B[index] - A[index];
    }
}

int main() {
    int filas = 4;
    int columnas = 4;
    int size = filas * columnas * sizeof(int);

    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);
    int *h_C = (int *)malloc(size);

    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j++) {
            h_A[i * columnas + j] = i * columnas + j + 1;
            h_B[i * columnas + j] = (i * columnas + j + 1) * 2;
        }
    }

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);


    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


    dim3 blockDim(2, 2);

    dim3 gridDim((columnas + blockDim.x - 1) / blockDim.x, 
                (filas + blockDim.y - 1) / blockDim.y);




    restarMatrices<<<gridDim, blockDim>>>(d_A, d_B, d_C, filas, columnas);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);



    printf("Matriz A:\n");
    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j++) {
            printf("%d ", h_A[i * columnas + j]);
        }
        printf("\n");
    }



    printf("\nMatriz B:\n");
    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j++) {
            printf("%d ", h_B[i * columnas + j]);
        }
        printf("\n");
    }



    printf("\nResultado (B - A):\n");
    for (int i = 0; i < filas; i++) {
        for (int j = 0; j < columnas; j++) {
            printf("%d ", h_C[i * columnas + j]);
        }
        printf("\n");
    }




    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
