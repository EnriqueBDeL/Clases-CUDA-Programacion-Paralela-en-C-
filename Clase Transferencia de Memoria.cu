#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


int main(int argc, char **argv){

//host = CPU + memoria principal(RAM)

        int n = 10;


        float *h_dato = (float*)malloc(sizeof(float)*n);



        for ( int i = 0; i < n; i++){

                h_dato[i] = i * 1;

        }


        printf("%nDatos del host:%n");
        for(int i = 0; i < n; i++){

                printf("h_dato[%d] = %f%n",i,h_dato[i]);

        }


//------------------------------------------------------------------------------------------------------------------------------------|

        float *d_dato;

        hipMalloc((void**)&d_dato,sizeof(float) * n); // "cudaMalloc" reserva memoria en la GPU.



        hipMemcpy(d_dato, h_dato, sizeof(float) * n, hipMemcpyHostToDevice); // "cudaMemcpy" copia datos entre CPU y la GPU.
        // "cudaMemcpyHostToDevice" indica que los datos deben copiarse desde el host a la memoria del dispositivo.


//------------------------------------------------------------------------------------------------------------------------------------|
//GPU

        float *h_resultado = (float*)malloc(sizeof(float) * n);

        hipMemcpy(h_resultado,d_dato,sizeof(float) * n, hipMemcpyDeviceToHost);
        // "cudaMemcpyDeviceToHost" indica que los datos deben copiarse desde la memoria del dispositivo a la memoria del host.

//------------------------------------------------------------------------------------------------------------------------------------|
//host


        printf("%nDatos depues de la copia a la GPU y de vuelta al host:%n");
        for(int i = 0; i < n; i++){

                printf("h_resultado[%d] = %f%n",i,h_resultado[i]);

        }


//Liberar memoria ----|
free(h_dato);
free(h_resultado);
hipFree(d_dato); //Libera la memoria que le fue asignada a la GPU mediante "cudaMalloc".
//--------------------|


return 0;
}
                              

