#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void sumaVectores(int *A, int *B, int *C, int N) {
  
  	int idx = blockIdx.x * blockDim.x + threadIdx.x;
   
 	if (idx < N) {

       		C[idx] = A[idx] + B[idx];

    	}
}



int main(int argc, char **argv) {


    int N = 16;
    int size = N * sizeof(int);

    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);
    int *h_C = (int *)malloc(size);



    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }



    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);


    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


    dim3 blockDim(4); // Establece que cada bloque contiene 4 hilos. 

    dim3 gridDim((N + 3) / 4); // Calcula cuántos bloques son necesarios para cubrir todos los elementos de "N"



    sumaVectores<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);



    printf("Vector A: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_A[i]);
    }
    printf("\n");



    printf("Vector B: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_B[i]);
    }
    printf("\n");



    printf("Resultado (A + B): ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_C[i]);
    }
    printf("\n");



    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
