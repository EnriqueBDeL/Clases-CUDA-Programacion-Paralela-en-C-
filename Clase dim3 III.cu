
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sumaMatrices3D(int *A, int *B, int *C, int z, int y, int x) {
   
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    if (idx < x && idy < y && idz < z) {

        int index = idz * y * x + idy * x + idx;

        C[index] = A[index] + B[index];
    }
}

int main() {


    int x = 3, y = 3, z = 3;
    int size = x * y * z * sizeof(int);


    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);
    int *h_C = (int *)malloc(size);


    for (int i = 0; i < z; i++) {
        for (int j = 0; j < y; j++) {
            for (int k = 0; k < x; k++) {
                h_A[i * y * x + j * x + k] = i * y * x + j * x + k + 1;
                h_B[i * y * x + j * x + k] = (i * y * x + j * x + k + 1) * 2;
            }
        }
    }



    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);


    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);



    dim3 blockDim(2, 2, 2);
    dim3 gridDim((x + blockDim.x - 1) / blockDim.x, 
                 (y + blockDim.y - 1) / blockDim.y, 
                 (z + blockDim.z - 1) / blockDim.z);



    sumaMatrices3D<<<gridDim, blockDim>>>(d_A, d_B, d_C, z, y, x);
    hipDeviceSynchronize();



    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);



    printf("Matriz A:\n");
    for (int i = 0; i < z; i++) {
        for (int j = 0; j < y; j++) {
            for (int k = 0; k < x; k++) {
                printf("%d ", h_A[i * y * x + j * x + k]);
            }
            printf("\n");
        }
        printf("\n");
    }




    printf("Matriz B:\n");
    for (int i = 0; i < z; i++) {
        for (int j = 0; j < y; j++) {
            for (int k = 0; k < x; k++) {
                printf("%d ", h_B[i * y * x + j * x + k]);
            }
            printf("\n");
        }
        printf("\n");
    }




    printf("Resultado (A + B):\n");
    for (int i = 0; i < z; i++) {
        for (int j = 0; j < y; j++) {
            for (int k = 0; k < x; k++) {
                printf("%d ", h_C[i * y * x + j * x + k]);
            }
            printf("\n");
        }
        printf("\n");
    }



    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
