/*

CUDA (Compute Unified Device Architecture) es una plataforma de computación paralela desarrollada por NVIDIA. 
Permite a los desarrolladores utilizar las GPU (Unidades de Procesamiento Gráfico) para realizar cálculos de propósito general, no solo gráficos.

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>//Contiene definiciones y funciones necesarias para programar y gestionar aplicaciones CUDA en C.

int main (int argc, char **argv){

        int contar;
        hipGetDeviceCount(&contar);

        printf("\nEl numero de GPUs disponibles es: %d\n\n",contar);



}
